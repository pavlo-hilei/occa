
#include <hip/hip_runtime.h>
#define block_size 32
extern "C" __global__ __launch_bounds__(block_size) void reduce(
  float *g_idata,
  float *res)
{
 {
   int bid = (0) + blockIdx.x;
   extern __shared__ float sdata[];
   {
     int tid = (0) + threadIdx.x;
     int i = bid * block_size + tid;
     sdata[tid] = g_idata[i];
     __syncthreads();
     for (unsigned int s = block_size / 2; s > 0; s >>= 1) {
       if (tid < s) {
         sdata[tid] += sdata[tid + s];
       }
       __syncthreads();
     }
     if (tid == 0)
       atomicAdd(res, sdata[0]);
   }
 }
}